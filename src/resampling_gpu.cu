#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include "resampling_gpu.h"

#include "scanFan_no_mm.h"

/*
 * Normalize a scanned array
 * The result array elements are in the range of [0, N] (not [0,1])
 */
template <typename T>
__global__ void normalize(T *array_new, T *array, size_t N)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ T sum;
    if(threadIdx.x == 0){
        sum = array[N-1];
        //printf(" thread %d is loading %e  N is %d\n", (int)tid, (double)sum, (int) N);
    }
    __syncthreads();
    if (tid < N)
        array_new[tid] = array[tid]/sum*(T)N;
}

/*
 * Search sp in array_normalized (also sorted) using binary search
 */
template <typename T>
__device__ int binSearch(T *array_normalized, size_t N, T sp)
{
    if (sp <= array_normalized[0])
        return 0;
    int low = 0;
    int high = N-1;
    int mid;
    while (low + 1 < high){
        mid = (low + high)/2;
        if (sp <= array_normalized[mid])
            high = mid;
        else
            low = mid;
    }
    return high;
}

/*
 * Pick N indices to represent the sampled elements.
 */
template <typename T>
__global__ void select(int *indexA, T *array_normalized, size_t N, T rand_mu)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    T sp = (T)tid + rand_mu;
    if (tid < N)
        indexA[tid] = binSearch<T>(array_normalized, N, sp);
}

/*
 * Update particles with the picked indices
 */
template <typename T>
__global__ void update(T *particles_new, T *particles, int *indexA, size_t N, size_t sDim)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    T tempVal;
    if (tid < N)
        for (int i = 0 ; i < sDim ; i ++){
            tempVal = particles[i*N + indexA[tid]];
            particles_new[tid + i*N] = tempVal;
    }
}

/*
 * This function wraps the CUDA kernels that perform systematic resampling on particles
 * The result is saved in particles_new.
 */
// template <typename T, typename T_data>
// int systematic_resample_kernel(T_data *particles_new, T_data *particles, size_t sDim,
//                                T *weight_scanned,
//                                size_t N, size_t block_size,
//                                T rand_mu)
// {
//     T *weight_scanned_normalized;
//     checkCUDAError(hipMalloc((void**) &weight_scanned_normalized, sizeof(T)*N));

//     int *indexA;
//     checkCUDAError(hipMalloc((void**) &indexA, sizeof(int)*N));

//     normalize<T><<<N/block_size, block_size>>>(weight_scanned_normalized, weight_scanned, N);
//     select<T><<<N/block_size, block_size>>>(indexA, weight_scanned_normalized, N, rand_mu);
//     update<T_data><<<N/block_size, block_size>>>(particles_new, particles, indexA, N, sDim);

//     checkCUDAError(hipFree(weight_scanned_normalized));
//     checkCUDAError(hipFree(indexA));

//     return 0;
// }

ResamplingGPU::ResamplingGPU(int NP, int dim):
    NP_(NP), dim_(dim)
{
    /* allocate space for partilces and its copy */
    unsigned int pp_sz = sizeof(float) * NP_ * dim_;
    checkCUDAError(hipMalloc((void**)&d_particles_, pp_sz));
    checkCUDAError(hipMalloc((void**)&d_particles_new_, pp_sz));

    /* allocate space for weight and normalized weight */
    unsigned int w_sz = sizeof(double) * NP_;
    checkCUDAError(hipMalloc((void**)&d_w_, w_sz));
    checkCUDAError(hipMalloc((void**)&d_w_normalized_, w_sz));

    /* allocate space for g_scan */
    level_ = 0;
    int numEle = NP_;
    if(NP_ <= NUM_THREADS)
        g_scan_ = nullptr;
    else{
        while(numEle > NUM_THREADS){
            level_ ++;
            numEle = numEle / NUM_THREADS;
        }
        g_scan_ = (double**) malloc(level_ * sizeof(double*));
        numEle = NP_;
        level_ = 0;
        while(numEle > NUM_THREADS){
            numEle = numEle / NUM_THREADS;
            checkCUDAError(hipMalloc((void**) &g_scan_[level_],  numEle * sizeof(double)));
            level_ ++;
        }
    }

    /* allocate space for indexA */
    checkCUDAError(hipMalloc((void**) &d_indexA_, sizeof(int)*NP_));
}

ResamplingGPU::~ ResamplingGPU()
{
    checkCUDAError(hipFree(d_particles_));
    checkCUDAError(hipFree(d_particles_new_));
    checkCUDAError(hipFree(d_w_));
    checkCUDAError(hipFree(d_w_normalized_));
    checkCUDAError(hipFree(d_indexA_));
    if (NP_ > NUM_THREADS){
        for (int i = 0; i < level_; i++)
            checkCUDAError(hipFree(g_scan_[i]));
        free((void**)g_scan_);
    }
}

void ResamplingGPU::setParticles(float *px, float *py, float *pz)
{
    unsigned int pp_sz = sizeof(float) * NP_;
    checkCUDAError(hipMemcpy(d_particles_, px, pp_sz, H2D));
    checkCUDAError(hipMemcpy(d_particles_ + NP_, py, pp_sz, H2D));
    checkCUDAError(hipMemcpy(d_particles_ + 2*NP_, pz, pp_sz, H2D));
}

void ResamplingGPU::getParticles(float *px, float *py, float *pz)
{
    unsigned int pp_sz = sizeof(float) * NP_;
    checkCUDAError(hipMemcpy(px, d_particles_new_, pp_sz, D2H));
    checkCUDAError(hipMemcpy(py, d_particles_new_ + NP_, pp_sz, D2H));
    checkCUDAError(hipMemcpy(pz, d_particles_new_ + 2*NP_, pp_sz, D2H));
}

void ResamplingGPU::setWeights(double *w)
{
    unsigned int w_sz = sizeof(double) * NP_;
    checkCUDAError(hipMemcpy(d_w_, w, w_sz, H2D));
}

void ResamplingGPU::getWeights(double *w)
{
    unsigned int w_sz = sizeof(double) * NP_;
    checkCUDAError(hipMemcpy(w, d_w_, w_sz, D2H));
}

void ResamplingGPU::doSystematicRes(float *px, float *py, float *pz, double *w)
{
    //printf("Calling gpu resampling ... \n");
    /*
     * 1. Allocate device memory for particles and its copies, weight,
     *    and some helper arrays
     * 2. Copy particles and weights to device memory
     * 3. Call scanFan_no_mm to perform prefix scan on the weight array
     * 4. Call systematicRes
     */

    /* Step 1: memory allocation */
    /* Done at constructor */

    /* Step 2: copy data from host to device */
    setWeights(w);
    // printf("  normalized weights [0-1]:\n");
    // for (int i = 0; i < NP_; i++){
    //     printf("%e  ", w[i]);
    //     if (i && (i+1)%10 == 0) printf("\n");
    // }
    /* pack px, py, and pangle into continuous memory d_particles */
    setParticles(px, py, pz);

    /* Step 3: perform prefix scan on the weights */
    scanFan_no_mm(d_w_, d_w_, NP_, NUM_THREADS, g_scan_);

    // getWeights(w);
    // printf("  scanned normalized weights:\n");
    // for (int i = 0; i < NP_; i++){
    //     printf("%e  ", w[i]);
    //     if (i && (i+1)%10 == 0) printf("\n");
    // }

    /* Step 4: perform systematic resampling */
    srand(time(NULL));
    double rand_mu = (double)rand() / (double)RAND_MAX;
    unsigned int blocks = (NP_ + NUM_THREADS - 1 ) / NUM_THREADS;
    //printf("  NP %d  blocks %d\n", NP_, blocks);
    normalize<<<blocks, NUM_THREADS>>>(d_w_normalized_, d_w_, NP_);

    // double *we = (double*)malloc(sizeof(double)*NP_);
    // hipMemcpy(we, d_w_normalized_, sizeof(double)*NP_, D2H);
    // printf("  scanned normalized weights [0-N]:\n");
    // for (int i = 0; i < NP_; i++){
    //     printf("%e  ", we[i]);
    //     if (i && (i+1)%10 == 0) printf("\n");
    // }


    select<<<blocks, NUM_THREADS>>>(d_indexA_, d_w_normalized_, NP_, rand_mu);

    // int *indexA = (int*)malloc(sizeof(int)*NP_);
    // hipMemcpy(indexA, d_indexA_, sizeof(int)*NP_, D2H);
    // printf("  indexA:\n");
    // for (int i = 0; i < NP_; i++){
    //     printf("%3d  ", indexA[i]);
    //     if (i && (i+1)%20 == 0) printf("\n");
    // }
    update<<<blocks, NUM_THREADS>>>(d_particles_new_, d_particles_, d_indexA_, NP_, dim_);

    /* Step 5: copy result back to host */
    getParticles(px, py, pz);
}